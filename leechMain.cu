#include "hip/hip_runtime.h"
#include <algorithm>
#include <array>
#include <cassert>
#define _USE_MATH_DEFINES
#include <cmath>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <random>
#include <sstream>
#include <vector>

using namespace std;
//using namespace chrono;

#include <tbb/parallel_for.h>

using Z = ptrdiff_t;
using R = float;

constexpr Z D = 24;
constexpr Z N = 196560;

constexpr R minE = -342.961657023412635055311941451;
constexpr R maxE = N;

constexpr R KNEEP = 69;//100;
constexpr R KNEEM = 2./3;

#include "codec.h"
#include "color.h"

using  ZV = array <Z , D>;
using  RV = array <R , D>;
using RVP = array <RV, 2>;

#include "leechData.h"

constexpr Z W = 1980;
constexpr Z H = 1080;
constexpr Z P = H*W;

constexpr R RES = 13;
constexpr const char* const FN = "medium5.mp4";

constexpr R XO = W/2;
constexpr R YO = H/2;

#include "leech.cu"

constexpr Z FS = 360;
constexpr R EPSILON = 1e-9;
constexpr Z SEED = 3;

ZV Vs[N/2];

#include "vectors.h"

void initLeech();

int main(int argc, const char* argv[]) {
    RGBEncoder encoder(FN, W, H, 30, 1);

    preprocess();
    initLeech();
    initCUDA();

    stringstream ss;

	//high_resolution_clock::time_point ot;
	//Z oy;

	//Z yp = -1;

    R* const data = new R[P];
	   RGB* const rgbFrame = new RGB   [P  ];
    //YUV420* const yuvFrame = new YUV420[P/4];

    for (Z n = 0; n < XY.size(); ++n)
        for (Z f = 0; f < FS; ++f) {
            const Z ff = FS*n + f;

            cerr << n << ' ' << f << ' ' << ff << endl;

            RVP xy = interpolate(XY[n], XY[n + 1], f);

            evaluateFrame(data, xy);

            for (int i = 0; i < P; i++)
                rgbFrame[i] = color(data[i] );

            /*
            tbb::parallel_for(
                //execution::par_unseq,
                tbb::blocked_range <int>  (0, H),
                [&] (tbb::blocked_range <int> r) {

            	for (Z y = r.begin(); y < r.end(); ++y) {
                    cout << y << endl;

            		const R yf = (y - YO)/RES;

            		for (Z x = 0; x < W; ++x) {
            			const R xf = (x - XO)/RES;

                        const RV z = mix(xf, xy[0], yf, xy[1] );

                        R sum = 0;

            			for (const ZV& r :  Vs)
            				sum += cosf(dot(r, z));

                        sum *= 2;

                        rgbFrame[W*y + x] = color(sum);
            		}
            	}}
            );
            */

            //rgb2yuv420(W, H, rgbFrame, yuvFrame);

            encoder.writeFrame(rgbFrame);
        }

    delete [] data, rgbFrame;//, yuvFrame;
}


Z golay[1 << 12];
Z ternary[759];

void initLeech() {
	Z count = 0, count1 = 0, count2 = 0;

	for (Z i = 0; i < 1 << 24; i++) {
		const bool parity1 = __builtin_popcount(i & 0x00000f) % 2;
		const bool parity2 = __builtin_popcount(i & 0x0000f0) % 2;
		const bool parity3 = __builtin_popcount(i & 0x000f00) % 2;
		const bool parity4 = __builtin_popcount(i & 0x00f000) % 2;
		const bool parity5 = __builtin_popcount(i & 0x0f0000) % 2;
		const bool parity6 = __builtin_popcount(i & 0xf00000) % 2;
		const bool parity7 = __builtin_popcount(i & 0x111111) % 2;

		constexpr Z W = 2;
		constexpr Z B = 3;

		constexpr Z WORDS[8] = {0, 1, W, B, B, W, 1, 0};
		Z words[6];

		words[0] = WORDS[(i & 0x00000e) >> 1];
		words[1] = WORDS[(i & 0x0000e0) >> 5];
		words[2] = WORDS[(i & 0x000e00) >> 9];
		words[3] = WORDS[(i & 0x00e000) >> 13];
		words[4] = WORDS[(i & 0x0e0000) >> 17];
		words[5] = WORDS[(i & 0xe00000) >> 21];

		constexpr Z HEXES[64] [6] = {
			{0, 0, 0, 0, 0, 0},


			{1, 1, W, W, B, B},
			{W, W, B, B, 1, 1},
			{B, B, 1, 1, W, W},

			{1, 1, B, B, W, W},
			{W, W, 1, 1, B, B},
			{B, B, W, W, 1, 1},


			{0, 0, 1, 1, 1, 1},
			{0, 0, W, W, W, W},
			{0, 0, B, B, B, B},

			{1, 1, 0, 0, 1, 1},
			{W, W, 0, 0, W, W},
			{B, B, 0, 0, B, B},

			{1, 1, 1, 1, 0, 0},
			{W, W, W, W, 0, 0},
			{B, B, B, B, 0, 0},


			{W, B, W, B, W, B},
			{B, 1, B, 1, B, 1},
			{1, W, 1, W, 1, W},

			{W, B, B, W, B, W},
			{B, 1, 1, B, 1, B},
			{1, W, W, 1, W, 1},

			{B, W, W, B, B, W},
			{1, B, B, 1, 1, B},
			{W, 1, 1, W, W, 1},

			{B, W, B, W, W, B},
			{1, B, 1, B, B, 1},
			{W, 1, W, 1, 1, W},



			{0, 1, 0, 1, W, B},
			{0, W, 0, W, B, 1},
			{0, B, 0, B, 1, W},

			{0, 1, W, B, 0, 1},
			{0, W, B, 1, 0, W},
			{0, B, 1, W, 0, B},

			{W, B, 0, 1, 0, 1},
			{B, 1, 0, W, 0, W},
			{1, W, 0, B, 0, B},


			{0, 1, 1, 0, B, W},
			{0, W, W, 0, 1, B},
			{0, B, B, 0, W, 1},

			{0, 1, B, W, 1, 0},
			{0, W, 1, B, W, 0},
			{0, B, W, 1, B, 0},

			{W, B, 1, 0, 1, 0},
			{B, 1, W, 0, W, 0},
			{1, W, B, 0, B, 0},


			{1, 0, 0, 1, B, W},
			{W, 0, 0, W, 1, B},
			{B, 0, 0, B, W, 1},

			{1, 0, W, B, 1, 0},
			{W, 0, B, 1, W, 0},
			{B, 0, 1, W, B, 0},

			{B, W, 0, 1, 1, 0},
			{1, B, 0, W, W, 0},
			{W, 1, 0, B, B, 0},


			{1, 0, 1, 0, W, B},
			{W, 0, W, 0, B, 1},
			{B, 0, B, 0, 1, W},

			{1, 0, B, W, 0, 1},
			{W, 0, 1, B, 0, W},
			{B, 0, W, 1, 0, B},

			{B, W, 1, 0, 0, 1},
			{1, B, W, 0, 0, W},
			{W, 1, B, 0, 0, B},
		};

		if (!(
				(parity1 && parity2 && parity3 && parity4 && parity5 && parity6 && parity7)
			||
				(!parity1 && !parity2 && !parity3 && !parity4 && !parity5 && !parity6 && !parity7)))

			goto fail;

		for (Z j = 0; j < 64; j++) {
			for (Z k = 0; k < 6; k++)
				if (words[k] != HEXES[j] [k])  goto loop;

			goto win;

			loop:;
		}

		goto fail;

		win:

		golay[count1++] = i;

		if (__builtin_popcount(i) == 8)
			ternary[count2++] = i;

    		fail:;
	}

	for (Z i = 0; i < 24; i++)
		for (Z j = 0; j < i; j++) {
			Vs[count] [j] = 4;
			Vs[count++] [i] = 4;

			Vs[count] [j] = 4;
			Vs[count++] [i] = -4;
		}

	for (Z i = 0; i < 759; i++) {
		for (Z signs = 0; signs < 256; signs += 2) {
			count2 = 0;

			if (!__builtin_parity(signs) % 2) {
				for (Z j = 0; j < 24; j++)
					if (ternary[i] & (1 << j))
						Vs[count] [j] = signs & (1 << count2++) ? -2 : 2;

				count++;
			}
		}
	}

	for (Z k = 0; k < 4096; k += 1) {
		for (Z i = 0; i < 24 && count != 98280; i++) {
			for (Z j = 0; j < 24; j++)
				Vs[count] [j] = 1;

			Vs[count] [i] = -3;

			for (Z l = 0; l < 24; l++)
				if (golay[k] & (1 << l))
					Vs[count] [l] = -Vs[count] [l];

			if (Vs[count] [0] > 0)
			   count++;
		}
	}
}
