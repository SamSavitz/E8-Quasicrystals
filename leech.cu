#include "hip/hip_runtime.h"
#include <algorithm>
#include <iomanip>
#include <iostream>
#include <random>
using namespace std;

//constexpr unsigned int N = 196560;
//constexpr unsigned char D = 24;

constexpr unsigned short HG = 1 << 11;

#include "golay.h"
#include "signs.h"

__constant__ unsigned int  CGOLAY[HG    ];
__constant__ unsigned char CSIGNS[1 << 6];


__global__ void evaluate(R* const out, const RVP xy, const int frame) {
	const auto tid = threadIdx.x;
	const auto id = 512*blockIdx.x + tid;

    const auto y = id/W;
    const auto x = id % W;

    if (y >= H || x >= W)
        return;

    const R xf = (x - XO)/RES;
    const R yf = (y - YO)/RES;

	__shared__ unsigned int  golay[HG    ];
	__shared__ unsigned char signs[1 << 6];

	golay[tid        ] = CGOLAY[tid        ];
	golay[tid +   512] = CGOLAY[tid +   512];
	golay[tid + 2*512] = CGOLAY[tid + 2*512];
	golay[tid + 3*512] = CGOLAY[tid + 3*512];

	if (tid < 1 << 6)
		signs[tid] = CSIGNS[tid];

	__syncthreads();

	R coords[D];

	#pragma unroll
	for (unsigned char d = 0; d < D; d++)
		  coords[d] = xf*xy[0] [d] + yf*xy[1] [d];

	R s = 0;

	for (unsigned char i = 1; i < D; i++)
		for (unsigned char j = 0; j < D; j++)
			if (j < i) {
				s += cosf(4*(coords[i] + coords[j] ));
				s += cosf(4*(coords[i] - coords[j] ));
			}

	for (unsigned char sign:  signs)
		for (unsigned short i = 0; i < 759; i++) {
			unsigned int octad = golay[i];
			unsigned char l = __ffs(octad) - 1;
			octad -= 1 << l;

			R a = coords[l];

			#pragma unroll
			for (signed char d = 1; d < 8; d++) {
				l = __ffs(octad) - 1;
				octad -= 1 << l;

				a += (((sign >> d) & 1) ? -1 : 1)*coords[l];
			}

			s += cosf(2*a);
		}

	for (const unsigned int cw:  golay) {
		R a = 0;

		#pragma unroll
		for (unsigned char i = 0; i < D; i++)
			a += (((cw >> i) & 1) ? 1 : -1)*coords[i];

		#pragma unroll
		for (unsigned char t = 0; t < D; t++)
			s += cosf((a + (((cw >> t) & 1) ? -4 : 4)*coords[t] ));
	}

	out[id] = 2*s;

	//out[id] = (maxE*((2*x + 30*frame) % W))/W;
}

R* deviceOutput;

void initCUDA() {
    hipDeviceSetCacheConfig(hipFuncCachePreferL1);

	hipMemcpyToSymbol(HIP_SYMBOL(CGOLAY), GOLAY, sizeof(GOLAY));
	hipMemcpyToSymbol(HIP_SYMBOL(CSIGNS), SIGNS, sizeof(SIGNS));

	hipMalloc(&deviceOutput, sizeof(R)*P);
}

void evaluateFrame(R* const data, const RVP& xy) {
	constexpr dim3 GRID_SIZE((P + 511)/512);
	constexpr dim3 BLOCK_SIZE(512);

	static int frame = 0;

	evaluate <<<GRID_SIZE, BLOCK_SIZE>>> (deviceOutput, xy, frame++);

     hipError_t err = hipGetLastError();

     if ( err != hipSuccess )
     {
        printf("CUDA Error: %s\n", hipGetErrorString(err));       

        // Possibly: exit(-1) if program cannot continue....
     }

	hipMemcpy(data, deviceOutput, sizeof(R)*P, hipMemcpyDeviceToHost);

	cout << data[0] << endl;
}
